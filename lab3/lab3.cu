#include "hip/hip_runtime.h"
#include "lab3.h"
#include <cstdio>

__device__ __host__ int CeilDiv(int a, int b) { return (a-1)/b + 1; }
__device__ __host__ int CeilAlign(int a, int b) { return CeilDiv(a, b) * b; }

__global__ void SimpleClone(
	const float *background,
	const float *target,
	const float *mask,
	float *output,
	const int wb, const int hb, const int wt, const int ht,
	const int oy, const int ox
)
{
	const int yt = blockIdx.y * blockDim.y + threadIdx.y;
	const int xt = blockIdx.x * blockDim.x + threadIdx.x;
	const int curt = wt*yt+xt;
	if (yt < ht and xt < wt and mask[curt] > 127.0f) {
		const int yb = oy+yt, xb = ox+xt;
		const int curb = wb*yb+xb;
		if (0 <= yb and yb < hb and 0 <= xb and xb < wb) {
			output[curb*3+0] = target[curt*3+0];
			output[curb*3+1] = target[curt*3+1];
			output[curb*3+2] = target[curt*3+2];
		}
	}
}

__global__ void CalculateFixed(
	const float *background,
	const float *target,
	const float *mask,
	float *fixed,
	const int wb, const int hb, const int wt, const int ht,
	const int oy, const int ox
)
{
	const int yt = blockIdx.y * blockDim.y + threadIdx.y;
	const int xt = blockIdx.x * blockDim.x + threadIdx.x;
	const int yb = oy+yt, xb = ox+xt;
	if (yt < ht and xt < wt and 0 <= yb and yb < hb and 0 <= xb and xb < wb) {
		const int curb = wb*yb+xb;
		const int curt = wt*yt+xt;
		fixed[curt * 3 + 0] = 0.0f;
		fixed[curt * 3 + 1] = 0.0f;
		fixed[curt * 3 + 2] = 0.0f;
		//N
		if(yt - 1 >= 0){
			const int curTargetId = wt * (yt - 1) + xt;
			fixed[curt * 3 + 0] += target[curt * 3 + 0] - target[curTargetId * 3 + 0];
			fixed[curt * 3 + 1] += target[curt * 3 + 1] - target[curTargetId * 3 + 1];
			fixed[curt * 3 + 2] += target[curt * 3 + 2] - target[curTargetId * 3 + 2];
			if(mask[curTargetId] < 127.0f and yb - 1 >= 0){
				const int curBackgroundId = wb * (yb - 1) + xb;
				fixed[curt * 3 + 0] += background[curBackgroundId * 3 + 0];
				fixed[curt * 3 + 1] += background[curBackgroundId * 3 + 1];
				fixed[curt * 3 + 2] += background[curBackgroundId * 3 + 2];
			}
		}
		//W
		if(xt - 1 >= 0){
			const int curTargetId = wt * yt + (xt - 1);
			fixed[curt * 3 + 0] += target[curt * 3 + 0] - target[curTargetId * 3 + 0];
			fixed[curt * 3 + 1] += target[curt * 3 + 1] - target[curTargetId * 3 + 1];
			fixed[curt * 3 + 2] += target[curt * 3 + 2] - target[curTargetId * 3 + 2];
			if(mask[curTargetId] < 127.0f and xb - 1 >= 0){
				const int curBackgroundId = wb * yb + (xb - 1);
				fixed[curt * 3 + 0] += background[curBackgroundId * 3 + 0];
				fixed[curt * 3 + 1] += background[curBackgroundId * 3 + 1];
				fixed[curt * 3 + 2] += background[curBackgroundId * 3 + 2];
			}
		}
		//S
		if(yt + 1 < ht){
			const int curTargetId = wt * (yt + 1) + xt;
			fixed[curt * 3 + 0] += target[curt * 3 + 0] - target[curTargetId * 3 + 0];
			fixed[curt * 3 + 1] += target[curt * 3 + 1] - target[curTargetId * 3 + 1];
			fixed[curt * 3 + 2] += target[curt * 3 + 2] - target[curTargetId * 3 + 2];
			if(mask[curTargetId] < 127.0f and yb + 1 < hb){
				const int curBackgroundId = wb * (yb + 1) + xb;
				fixed[curt * 3 + 0] += background[curBackgroundId * 3 + 0];
				fixed[curt * 3 + 1] += background[curBackgroundId * 3 + 1];
				fixed[curt * 3 + 2] += background[curBackgroundId * 3 + 2];
			}
		}
		//E
		if(xt + 1 < wt){
			const int curTargetId = wt * yt + (xt + 1);
			fixed[curt * 3 + 0] += target[curt * 3 + 0] - target[curTargetId * 3 + 0];
			fixed[curt * 3 + 1] += target[curt * 3 + 1] - target[curTargetId * 3 + 1];
			fixed[curt * 3 + 2] += target[curt * 3 + 2] - target[curTargetId * 3 + 2];
			if(mask[curTargetId] < 127.0f and xb + 1 < wb){
				const int curBackgroundId = wb * yb + (xb + 1);
				fixed[curt * 3 + 0] += background[curBackgroundId * 3 + 0];
				fixed[curt * 3 + 1] += background[curBackgroundId * 3 + 1];
				fixed[curt * 3 + 2] += background[curBackgroundId * 3 + 2];
			}
		}
	}
}

__global__ void PoissonImageCloningIteration(
	const float *fixed,
	const float *mask,
	const float *inputBuf,
	float *outputBuf,
	const int wt, const int ht
)
{
	const int yt = blockIdx.y * blockDim.y + threadIdx.y;
	const int xt = blockIdx.x * blockDim.x + threadIdx.x;
	if (yt < ht and xt < wt) {
		float fractionSum = 0;
		const int curt = wt*yt+xt;
		outputBuf[curt * 3 + 0] = fixed[curt * 3 + 0];
		outputBuf[curt * 3 + 1] = fixed[curt * 3 + 1];
		outputBuf[curt * 3 + 2] = fixed[curt * 3 + 2];

		//N
		if(yt - 1 >= 0){
			const int curTargetId = wt * (yt - 1) + xt;
			if(mask[curTargetId] > 127.0f){
				outputBuf[curt * 3 + 0] += inputBuf[curTargetId * 3 + 0];
				outputBuf[curt * 3 + 1] += inputBuf[curTargetId * 3 + 1];
				outputBuf[curt * 3 + 2] += inputBuf[curTargetId * 3 + 2];
			}
			fractionSum += 1.0f;
		}
		//W
		if(xt - 1 >= 0){
			const int curTargetId = wt * yt + (xt - 1);
			if(mask[curTargetId] > 127.0f){
				outputBuf[curt * 3 + 0] += inputBuf[curTargetId * 3 + 0];
				outputBuf[curt * 3 + 1] += inputBuf[curTargetId * 3 + 1];
				outputBuf[curt * 3 + 2] += inputBuf[curTargetId * 3 + 2];
			}
			fractionSum += 1.0f;
		}
		//S
		if(yt + 1 < ht){
			const int curTargetId = wt * (yt + 1) + xt;
			if(mask[curTargetId] > 127.0f){
				outputBuf[curt * 3 + 0] += inputBuf[curTargetId * 3 + 0];
				outputBuf[curt * 3 + 1] += inputBuf[curTargetId * 3 + 1];
				outputBuf[curt * 3 + 2] += inputBuf[curTargetId * 3 + 2];
			}
			fractionSum += 1.0f;
		}
		//E
		if(xt + 1 < wt){
			const int curTargetId = wt * yt + (xt + 1);
			if(mask[curTargetId] > 127.0f){
				outputBuf[curt * 3 + 0] += inputBuf[curTargetId * 3 + 0];
				outputBuf[curt * 3 + 1] += inputBuf[curTargetId * 3 + 1];
				outputBuf[curt * 3 + 2] += inputBuf[curTargetId * 3 + 2];
			}
			fractionSum += 1.0f;
		}
		//   answer/fractionSum
		outputBuf[curt * 3 + 0] /= fractionSum;
		outputBuf[curt * 3 + 1] /= fractionSum;
		outputBuf[curt * 3 + 2] /= fractionSum;
	}
}

void PoissonImageCloning(
	const float *background,
	const float *target,
	const float *mask,
	float *output,
	const int wb, const int hb, const int wt, const int ht,
	const int oy, const int ox
)
{
	// set up
	float *fixed, *buf1, *buf2;
	hipMalloc(&fixed, 3*wt*ht*sizeof(float));
	hipMalloc(&buf1, 3*wt*ht*sizeof(float));
	hipMalloc(&buf2, 3*wt*ht*sizeof(float));
	// initialize the iteration
	dim3 gdim(CeilDiv(wt,32), CeilDiv(ht,16)), bdim(32,16);
	CalculateFixed<<<gdim, bdim>>>(
		background, target, mask, fixed,
		wb, hb, wt, ht, oy, ox
	);
	hipMemcpy(buf1, target, sizeof(float)*3*wt*ht, hipMemcpyDeviceToDevice);
	// iterate
	for(int i=0;i<10000;++i){
		PoissonImageCloningIteration<<<gdim, bdim>>>(
	    	fixed, mask, buf1, buf2, wt, ht
	   	);
	   	PoissonImageCloningIteration<<<gdim, bdim>>>(
	    	fixed, mask, buf2, buf1, wt, ht
		);
	}
	//copy the image back
	hipMemcpy(output, background, wb*hb*sizeof(float)*3, hipMemcpyDeviceToDevice);
	SimpleClone<<<gdim, bdim>>>(
		background, buf1, mask, output,
		wb, hb, wt, ht, oy, ox
	);
	//clean up
	hipFree(fixed);
	hipFree(buf1);
	hipFree(buf2);
}