#include "hip/hip_runtime.h"
#include "counting.h"
#include <cstdio>
#include <cassert>
#include <thrust/scan.h>
#include <thrust/transform.h>
#include <thrust/functional.h>
#include <thrust/device_ptr.h>
#include <thrust/execution_policy.h>

const int NUM_THREAD = 1024;

__device__ __host__ int MyCeilDiv(int a, int b) { return (a-2)/b; }	//Index of hierarchy; input[0, n-1]; return [0, n-1]
__device__ __host__ int FloorDivWithTreeIndex(int a, int b) { return a*b + 2; }	//Index of tree; input[0, n-1]; return [0, n-1]
__device__ int FindIndexDevice(int height, int index) { return __double2int_rd(pow(2.0, __int2double_rn(height))) - 2 + index;}	//height starts from 1; index starts from 0

struct IsLetterUnary
{
  __host__ __device__
  int operator()(char x) { return x > '@' ? 1 : 0; }
};

__global__ void mapLetterToBit(const char* text, int *pos, int text_size){
	const int x = blockIdx.x * blockDim.x + threadIdx.x;
	if (x < text_size) {
		pos[x] = IsLetterUnary()(text[x]);
	}
}

__global__ void createTree(int* treePtr, int treeSize, int initPos, int length){
	const int x = blockIdx.x * blockDim.x + threadIdx.x;
	if (x < length) {
		int nextInitPos = FloorDivWithTreeIndex(initPos + x, 2); //Array input index start from 0
		if(treePtr[nextInitPos] == treePtr[nextInitPos + 1] && treePtr[nextInitPos] > 0){
			treePtr[initPos + x] = treePtr[nextInitPos + 1] + treePtr[nextInitPos];
		}
	}
}

__global__ void treeParsing(const int* treePtr, int* posOutput, int initPos, int length, int treeTotalHeight){
	const int x = blockIdx.x * blockDim.x + threadIdx.x;
	if (x < length) {
		int sum = 0;
		int height = treeTotalHeight;
		int nextIndex = initPos + x;
		while(height>1 && treePtr[nextIndex] >= 1 && nextIndex >= FindIndexDevice(height, 0)){
			sum += treePtr[nextIndex] * ((nextIndex + 1) % 2);
			nextIndex = MyCeilDiv(nextIndex, 2) - ((nextIndex + 1) % 2);
			height--;
		}
		if(nextIndex >= FindIndexDevice(height, 0)){
			while(height<=treeTotalHeight){
				if(treePtr[nextIndex] > 0){
					sum += treePtr[nextIndex];
					nextIndex = FloorDivWithTreeIndex(nextIndex, 2) - 1;
				}else{
					nextIndex = FloorDivWithTreeIndex(nextIndex, 2) + 1;
				}
				height++;
			}
		}
		posOutput[x] = sum;
	}
}

void CountPosition1(const char *text, int *pos, int text_size)
{
	thrust::transform(thrust::device, text, text + text_size, pos, IsLetterUnary());
	thrust::inclusive_scan_by_key(thrust::device, pos, pos + text_size, pos, pos, thrust::equal_to<int>());
}

void CountPosition2(const char *text, int *pos, int text_size)
{
	mapLetterToBit<<<(text_size-1)/NUM_THREAD + 1, NUM_THREAD>>>(text, pos, text_size);

	int treeHeight = (int)ceil(log((double)text_size) / log(2.0));

	int treeTotalSize = pow(2, treeHeight) * 2 - 2;
	int textOffset = pow(2, treeHeight) - 2;
	int *gpuTree;
	int *gpuPosOutput;
	hipMalloc((void**)&gpuTree, sizeof(int) * treeTotalSize);
	hipMemset(gpuTree, 0, sizeof(int) * treeTotalSize);
	hipMemcpy(gpuTree + textOffset, pos, sizeof(int) * text_size, hipMemcpyDeviceToDevice);
	hipMalloc((void**)&gpuPosOutput, sizeof(int) * text_size);
	hipMemset(gpuPosOutput, 0, sizeof(int) * text_size);
	for(int i = treeHeight - 1;i>=0;i--){
		int textOffset = pow(2, i) - 2;
		int length = pow(2, i);
		int blockSize = (length-1)/NUM_THREAD + 1;
		createTree<<<blockSize, NUM_THREAD>>>(gpuTree, treeTotalSize, textOffset, length);
	}

	treeParsing<<<(text_size-1)/NUM_THREAD + 1, NUM_THREAD>>>(gpuTree, gpuPosOutput, textOffset, text_size, treeHeight);

	hipMemcpy(pos, gpuPosOutput, sizeof(int) * text_size, hipMemcpyDeviceToDevice);

	hipFree(gpuTree);
	hipFree(gpuPosOutput);

}