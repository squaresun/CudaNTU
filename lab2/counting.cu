#include "hip/hip_runtime.h"
#include "counting.h"
#include <cstdio>
#include <cassert>
#include <thrust/scan.h>
#include <thrust/transform.h>
#include <thrust/functional.h>
#include <thrust/device_ptr.h>
#include <thrust/execution_policy.h>

////////////////////////////////////////////////////////////////////////////////////////////////////////////
#include <thrust/for_each.h>

struct printf_functor
{
  __host__ __device__
  void operator()(int x)
  {
    // note that using printf in a __device__ function requires
    // code compiled for a GPU with compute capability 2.0 or
    // higher (nvcc --arch=sm_20)
    printf("%d\n", x);
  }
  __host__ __device__
  void operator()(char x)
  {
    // note that using printf in a __device__ function requires
    // code compiled for a GPU with compute capability 2.0 or
    // higher (nvcc --arch=sm_20)
    printf("%c", x);
  }
};
////////////////////////////////////////////////////////////////////////////////////////////////////////////

const int NUM_THREAD = 1024;

__device__ __host__ int CeilDiv(int a, int b) { return (a-1)/b + 1; }
__device__ __host__ int CeilAlign(int a, int b) { return CeilDiv(a, b) * b; }

struct IsLetterUnary
{
  __host__ __device__
  int operator()(char x) { return x > '@' ? 1 : 0; }
};

__global__ void mapLetterToBit(const char* text, int *pos, int text_size){
	// const int y = blockIdx.y * blockDim.y + threadIdx.y;
	const int x = blockIdx.x * blockDim.x + threadIdx.x;
	if (x < text_size) {
		pos[x] = IsLetterUnary()(text[x]);
	}
}

__global__ void segmentedPrefixSum(const char* text, int *pos, int text_size){
	// const int y = blockIdx.y * blockDim.y + threadIdx.y;
	const int x = blockIdx.x * blockDim.x + threadIdx.x;
	if (x < text_size) {
		
	}
}

void CountPosition1(const char *text, int *pos, int text_size)
{
	thrust::transform(thrust::device, text, text + text_size, pos, IsLetterUnary());
	thrust::inclusive_scan_by_key(thrust::device, pos, pos + text_size, pos, pos, thrust::equal_to<int>());
}

void CountPosition2(const char *text, int *pos, int text_size)
{
	mapLetterToBit<<<(text_size-1)/NUM_THREAD + 1, NUM_THREAD>>>(text, pos, text_size);
	segmentedPrefixSum<<<(text_size-1)/NUM_THREAD + 1, NUM_THREAD>>>(text, pos, text_size);
}
