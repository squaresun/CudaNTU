#include "hip/hip_runtime.h"
#include <random>
#include <vector>
#include <tuple>
#include <cstdio>
#include <cstdlib>
#include <functional>
#include <algorithm>
#include "SyncedMemory.h"
#include "Timer.h"
#include "counting.h"
using namespace std;

#define CHECK {\
	auto e = hipDeviceSynchronize();\
	if (e != hipSuccess) {\
		printf("At " __FILE__ ":%d, %s\n", __LINE__, hipGetErrorString(e));\
		abort();\
	}\
}

template <typename Engine>
tuple<vector<char>, vector<int>> GenerateTestCase(Engine &eng, const int N) {
	poisson_distribution<int> pd(14.0);
	bernoulli_distribution bd(0.1);
	uniform_int_distribution<int> id1(1, 20);
	uniform_int_distribution<int> id2(1, 5);
	uniform_int_distribution<int> id3('a', 'z');
	tuple<vector<char>, vector<int>> ret;
	auto &text = get<0>(ret);
	auto &pos = get<1>(ret);
	auto gen_rand_word_len = [&] () -> int {
		return max(1, min(500, pd(eng) - 5 + (bd(eng) ? id1(eng)*20 : 0)));
	};
	auto gen_rand_space_len = [&] () -> int {
		return id2(eng);
	};
	auto gen_rand_char = [&] () {
		return id3(eng);
	};
	auto AddWord = [&] () {
		int n = gen_rand_word_len();
		for (int i = 0; i < n; ++i) {
			text.push_back(gen_rand_char());
			pos.push_back(i+1);
		}
	};
	auto AddSpace = [&] () {
		int n = gen_rand_space_len();
		for (int i = 0; i < n; ++i) {
			text.push_back('\n');
			pos.push_back(0);
		}
	};

	AddWord();
	while (text.size() < N) {
		AddSpace();
		AddWord();
	}
	return ret;
}

void TestRoutine(
	SyncedMemory<int>& yours_sync, SyncedMemory<char>& text_sync,
	const int n, const int part, const int *golden
) {
	// Initialization
	Timer timer_count_position;
	int *yours_gpu = yours_sync.get_gpu_wo();
	hipMemset(yours_gpu, 0, sizeof(int)*n);

	// Run
	timer_count_position.Start();
	if (part == 1) {
		CountPosition1(text_sync.get_gpu_ro(), yours_gpu, n);
	} else {
		CountPosition2(text_sync.get_gpu_ro(), yours_gpu, n);
	}
	CHECK;
	timer_count_position.Pause();

	// Part I check
	const int *yours = yours_sync.get_cpu_ro();
	int n_match = mismatch(golden, golden+n, yours).first - golden;

	printf_timer(timer_count_position);
	if (n_match != n) {
		printf("Part %d WA\n", part);
	} else {
		printf("Part %d AC\n", part);
	}
}

int main(int argc, char **argv)
{
	// Initialize random text
	default_random_engine engine(12345);
	auto text_pos_head = GenerateTestCase(engine, 40000000); // 40 MB data
	vector<char> &text = get<0>(text_pos_head);
	vector<int> &pos = get<1>(text_pos_head);

	// Prepare buffers
	int n = text.size();
	char *text_gpu;
	hipMalloc(&text_gpu, sizeof(char)*n);
	SyncedMemory<char> text_sync(text.data(), text_gpu, n);
	text_sync.get_cpu_wo(); // touch the cpu data
	MemoryBuffer<int> yours1_buf(n);
	MemoryBuffer<int> yours2_buf(n);
	auto yours1_mb = yours1_buf.CreateSync(n);
	auto yours2_mb = yours2_buf.CreateSync(n);

	// We test 2 in first to prevent cheating
	TestRoutine(yours1_mb, text_sync, n, 2, pos.data());
	TestRoutine(yours2_mb, text_sync, n, 1, pos.data());

	hipFree(text_gpu);
	return 0;
}
