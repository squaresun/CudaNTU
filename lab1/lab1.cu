#include "hip/hip_runtime.h"
#include "lab1.h"
static const unsigned W = 640;
static const unsigned H = 960;
static const unsigned NFRAME = 7200;
static const unsigned LANE1START = 120;
static const unsigned LANE4END = 520;
static const unsigned LANEMID = 320;
static const unsigned NUMBEROFTIMEQUEUE = 6;
static const unsigned TIMEQUEUEMAXSIZE = 400;
static const unsigned NOTEYLENGTH = 12;
static const unsigned CRITICALLINEYBOTTOM = 55;	//bottom
static const unsigned CRITICALLINEYTOP = 70;	//top
static const unsigned CRITICALLINEXLEFT = 100;
static const unsigned CRITICALLINEXRIGHT = 540;

static const int LANE1NOTEONTIMEARR[TIMEQUEUEMAXSIZE] = {0,724,1241,1655,2379,2896,3310,4034,4551,4965,5689,6206,7655,9310,9827,10965,11689,12620,13137,13965,14793,15724,15931,17275,18103,19034,19241,20482,21310,22137,22965,23275,23482,23689,23896,24103,24310,24517,24724,25034,25241,25448,26482,26689,26896,27103,27310,28137,28551,28965,29379,29793,30206,30620,31034,31448,31862,32275,32689,33103,33517,33827,34344,34655,35068,35379,35689,36000,36413,36827,37137,37655,37862,38379,38689,39000,39310,39620,39827,40034,40551,40758,41379,41586,42103,42517,42724,43448,44172,45310,46344,47275,47896,48724,49137,49344,50068,50793,51931,52965,53172,53931,54620,54827,55344,55758,56689,57000,57517,57931,58137,59379,59689,60000,60413,60827,61655,61965,63103,63827,64241,64551,64965,65793,66413,66827,67862,68275,68689,69103,70137,74482,74896,75310,75724,76551,76689,76827,76965,77379,78310,78517,79551,79758,79965,80172,80793,81000,81931,82344,82862,83172,83586,84103,84517,84827,85241,85862,86482,87000,87827,88448,88965,89482,89793,90206,90413,91344,91758,92275,92689,93000,93310,93931,94448,94655,95172,95586,96931,97241,97551,97862,98172,98482,100448,100862,101275,101896,102310,102517,102931,103137,103655,103965,104275,104793,105103,105931,106862,107068,107793,108413};
static const int LANE2NOTEONTIMEARR[TIMEQUEUEMAXSIZE] = {413,1034,1448,1758,2068,2689,3000,3206,3724,4344,4758,5068,5379,6000,6310,6517,7344,7965,8379,8793,9620,10655,11275,12103,12931,13551,14379,15000,15413,16862,17689,18310,18724,19137,19344,19655,20275,21103,21931,22758,23172,23379,23586,23793,24000,24206,24413,24620,25655,25862,26068,26275,26965,27724,28448,28758,29172,29586,29896,30103,30413,30827,31137,31758,32068,32482,32896,33413,33620,34448,34965,35172,35482,35896,36206,36517,36724,37034,37344,37551,37965,38275,38482,38793,39206,39413,39827,40034,40551,40758,41482,41689,42103,42413,42827,43344,43655,44068,44275,45310,45620,46241,46655,46862,47068,47482,47793,48000,48137,48275,48724,49034,49448,49965,50275,50689,50896,51310,51448,51586,51931,52241,52862,53068,53275,53862,54137,54413,55034,55448,56172,56379,57103,57413,57827,58655,58758,59586,60310,60724,61241,61448,62275,62586,63413,63931,64344,64758,65172,66103,66413,66827,67448,67862,68172,68275,68586,68689,69000,69103,69413,69620,70034,70241,70551,70862,71172,72965,73344,73758,74068,74275,74482,74793,74896,75206,75310,75517,75724,76034,76241,77172,77586,78724,78931,79137,79344,79965,80172,80379,80586,80793,81000,81517,82034,82241,82551,82758,83068,83482,83689,84000,84206,84413,84724,85137,85344,85758,85965,86379,86586,87103,87413,87620,87931,88344,89068,89379,89689,89896,90310,90517,90827,91137,91655,92379,92896,93103,94344,94551,95482,95793,96103,96517,96724,97137,97344,97758,98068,98275,98689,99103,99413,99827,100034,100344,100551,100758,100965,101172,101793,103551,103758,104172,104379,104586,104896,105310,106137,106758,106965,107586,107862,108206};
static const int LANE3NOTEONTIMEARR[TIMEQUEUEMAXSIZE] = {827,1344,1862,2482,3103,4137,4655,5172,5793,6413,7137,7758,9000,9413,9724,10448,11068,11793,12310,12724,13034,13862,14689,15103,15517,16241,16448,17172,18000,18413,18827,19655,20068,20896,21724,22551,23172,23379,23586,23793,24000,24206,24413,24620,24931,25137,25344,25551,26586,26793,27034,27206,27724,28241,28551,29068,29275,29482,29689,30206,30517,30931,31241,31551,31862,32379,32586,32793,33000,33310,33724,34034,34241,34551,34862,35275,35793,36103,36310,36620,36931,37448,37758,38172,38586,39103,39517,39724,39931,40655,40862,42000,42310,42931,43241,43758,43965,44793,45000,45413,45724,46137,46551,46758,46965,47172,47689,48000,48137,48275,48620,48931,49551,49862,50379,50586,51310,51448,51586,52034,52344,52758,53068,53275,53793,54068,54413,55137,55862,56482,57310,57724,58655,58758,59689,60000,60310,60620,61241,61448,62068,62689,63000,63620,64034,64448,64758,65275,66000,66517,66724,66931,67448,68172,68586,69000,69310,69517,69724,69931,70344,70655,70965,71586,71896,72206,72551,72689,72827,73103,73241,73655,73965,74172,74379,74793,75206,75517,75931,76137,76344,76551,76689,76827,77068,77482,78206,78413,78620,78827,79862,80068,80275,80482,80689,80896,81103,81517,82137,82448,82655,82965,83379,83793,84310,84620,85034,85448,85655,86068,86275,86689,86896,87206,87517,88034,88241,88655,88862,89172,89586,90000,90724,90931,91241,91551,91965,92172,92482,92793,93413,94344,94551,95379,95689,96000,96206,96620,97034,97448,97655,98586,98793,99000,99206,99620,99931,100241,100655,101068,101379,101586,102000,102206,102620,102827,103241,103448,105517,106344,106758,106965,107655,107931,108206};
static const int LANE4NOTEONTIMEARR[TIMEQUEUEMAXSIZE] = {0,206,517,1137,1655,2172,2793,3310,3517,3827,4448,4965,5482,6103,6620,6827,7448,8068,8482,9103,9931,10137,10758,11379,12413,13448,14275,15206,15620,16137,16344,16758,17586,18517,18931,19862,20689,21517,22344,23275,23482,23689,23896,24103,24310,24517,24724,25758,25965,26172,26379,28344,28655,28862,29379,30000,30310,30724,31344,31655,31965,32172,32689,33206,33931,34137,34758,35172,35586,36000,36413,36827,37241,37655,38068,38482,38896,39310,39724,39931,40655,40862,42000,43034,43862,44379,44689,44896,45413,45827,46034,47379,47586,48620,49655,50482,51000,52034,52448,52655,52965,53172,54000,54620,54827,55551,56068,56275,56793,57206,57620,57931,58137,59379,59793,60103,60620,60827,61758,62379,62896,63310,63724,64137,64551,64965,65793,66517,66724,66931,71379,71689,72000,72310,72551,72689,72827,73103,73241,73551,77275,77689,79034,79241,79448,79655,79862,80068,80689,80896,81103,83275,83896,84931,85551,86172,86793,87310,87724,88137,88551,88758,89275,90103,90620,91034,91448,91862,92068,92586,93206,93931,94448,94655,95275,95896,96310,96827,97965,98379,98896,99310,99517,100137,101482,101689,102103,102413,102724,103034,103344,103862,104068,104482,104689,105000,105724,106551,106862,107068,107724,108413};
static const int LANE5NOTEONTIMEARR[TIMEQUEUEMAXSIZE] = {6620,7034,7448,7862,8275,8689,9103,9517,9931,10344,10758,11172,11586,12000,12413,12827,13448,13862,14275,14689,15103,15517,16137,16758,17172,17586,18000,18413,18827,19448,19862,20068,20275,20482,20689,20896,21103,21310,21517,21724,21931,22137,22344,22551,22758,22965,24827,27724,28862,30517,32172,33931,40137,40965,41793,42620,43034,43862,44482,44689,45517,46551,48413,49241,49655,50482,51103,51310,52137,53379,54206,56172,56379,58344,58448,58965,59068,59172,59275,60827,63103,63413,63724,65379,65482,65586,65689,66206,66620,67034,68068,68482,68896,69517,69724,69931,70655,70965,74689,75103,75517,76137,76344,76551,78206,81517,85241,85655,86068,86482,86896,87310,87724,88137,88551,90620,91034,91448,91862,93517,93655,93793,93931,94758,101379,103448,105103,105310,105517,105724,105931,106137,106344,106551,107172,108000,108413};
static const int LANE6NOTEONTIMEARR[TIMEQUEUEMAXSIZE] = {13241,13655,14068,14482,14896,15310,15724,16551,16965,17379,17793,18206,18620,19034,19862,20068,20275,20482,20689,20896,21103,21310,21517,21724,21931,22137,22344,22551,22758,22965,24827,27724,33103,35482,37137,38793,40137,40965,41379,42206,43448,44068,44586,45103,45931,46344,48000,48827,50068,50689,51206,51724,52551,53379,54206,55137,55551,55862,56482,56793,57103,58344,58448,58965,59068,59172,59275,60827,61758,62068,62379,62689,65379,65482,65586,65689,66206,67448,71172,71586,71689,71896,72000,72206,72310,72965,73758,74068,77793,79448,81517,81931,82344,82758,83172,83586,84000,84413,84827,88965,89379,89793,90206,92275,92689,93103,93517,93655,93793,93931,95172,96413,99724,101793,105103,105310,105517,105724,105931,106137,106344,106551,107172,108000,108413};

static const int CAMTIMEARR[TIMEQUEUEMAXSIZE] = {0, 19862, 24827, 27310, 27724, 27854, 38068, 40137, 40267, 40965, 41095, 53379, 53509, 54206, 54336, 57103, 60827, 61241, 64551, 67034, 67448, 67861, 71172, 71585, 74482, 74895, 77793, 78206, 81103, 81517, 81620, 93931, 94034, 94759, 95172, 95586, 96414, 105103, 105103, 106759, 107172, 107275, 108000, 108103, 108414, 108517, 999999};
static const float CAMROTATEARR[TIMEQUEUEMAXSIZE] = {0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0.3, 0.3, -0.3, -0.3, 0.3, 0.3, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0};
static const float CAMSCALEARR[TIMEQUEUEMAXSIZE] = {1, 1, 1.6, 0.8, 0.8, 1, 1, 0.9, 1.1, 1.1, 1.3, 1.3, 1.1, 1.1, 0.9, 0.8, 0.8, 1, 1, 1.3, 1.3, 1.3, 1.3, 1.3, 1.3, 1.3, 1.3, 0.8, 0.8, 1, 1.1, 1.1, 0.9, 0.9, 1.6, 1.6, 1.6, 1.6, 1.6, 1.1, 1.1, 0.9, 0.9, 1, 1, 1};
static const int CAMTRAPEZOIDFACTORARR[TIMEQUEUEMAXSIZE] = {0, 0, 300, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 300, 300, 300, 300, 300, 300, 300, 300, 0, 0, 0, 0, 0, 0, 0, 300, 300, 300, 300, 300, 0, 0, 0, 0, 0, 0, 0};

static const int LANE1NOTEDURATIONTIMEARR[TIMEQUEUEMAXSIZE] = {0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,207,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,414,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,827,0,0,0,0,0,1242,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0};
static const int LANE2NOTEDURATIONTIMEARR[TIMEQUEUEMAXSIZE] = {0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,207,0,0,0,0,0,0,0,0,0,0,207,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,207,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,1448,0,0,0,0,0,0,0,0,0,0,0,0,0};
static const int LANE3NOTEDURATIONTIMEARR[TIMEQUEUEMAXSIZE] = {0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,207,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,620,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,1448,0,0,0,0,0,0,0};
static const int LANE4NOTEDURATIONTIMEARR[TIMEQUEUEMAXSIZE] = {0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,206,0,0,0,0,0,0,0,207,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,414,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,828,0,0,0,0,0,1242,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0};
static const int LANE5NOTEDURATIONTIMEARR[TIMEQUEUEMAXSIZE] = {0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,207,206,207,207,207,207,414,207,207,207,206,207,207,207,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,2483,0,517,517,517,413,0,0,207,0,207,206,0,621,827,621,207,0,207,207,0,621,828,0,0,0,0,0,0,0,0,0,0,414,0,0,827,0,0,0,0,207,0,414,207,207,207,0,0,413,0,0,207,207,207,0,0,414,2897,0,0,0,0,0,0,0,0,0,414,0,0,0,413,0,0,0,0,414,414,1448,0,0,0,0,0,0,0,0,0,0,0};
static const int LANE6NOTEDURATIONTIMEARR[TIMEQUEUEMAXSIZE] = {0,0,0,0,207,207,413,0,0,0,0,207,207,621,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,2483,0,414,518,518,517,0,0,621,828,207,207,0,207,0,207,620,828,207,207,0,207,0,0,0,0,0,0,0,0,828,0,0,0,0,0,0,414,0,0,0,0,0,0,0,0,207,0,207,0,0,0,0,0,0,0,0,414,413,1655,0,0,0,0,0,0,0,0,0,0,0,0,414,0,0,0,0,0,0,0,414,414,413,1448,0,0,0,0,0,0,0,0,0,0,0};

__device__ int noOfFrameOnDevice[1];
__device__ float fxDecayOnDevice[NUMBEROFTIMEQUEUE];
__device__ int timeQueueStartIndexArrOnDevice[NUMBEROFTIMEQUEUE];
__device__ float rotateAngleOnDevice[1];
__device__ float scaleFactorOnDevice[1];
__device__ int trapezoidFactorOnDevice[1];

template <typename T>
__host__ __device__
inline T lerp(T v0, T v1, T t) {
    return fma(t, v1, fma(-t, v0, v0));
}

struct NoteTimeQueue{
	int laneIndex;
	int endIndex;
	int noteOnTimeQueue[TIMEQUEUEMAXSIZE];
	int noteDurationTimeQueue[TIMEQUEUEMAXSIZE];
	__device__ bool isInsideNote(int y){
		for(int i = timeQueueStartIndexArrOnDevice[laneIndex];i<endIndex;i++){
			float delta = __int2float_rd(y) * expf(__int2float_rd(y)/__int2float_rd(W)) - __int2float_rd(noteOnTimeQueue[i] - __float2int_rd(__int2float_rd(noOfFrameOnDevice[0]) / 60.0 * 1000.0));
			if(delta > 0 and delta < __int2float_rd(noteDurationTimeQueue[i] > NOTEYLENGTH ? noteDurationTimeQueue[i] : NOTEYLENGTH)){
				return true;
			}
		}
		return false;
	}
};

struct Lab1VideoGenerator::Impl {
	int t = 0;
	float rotateAngle = 0.0;	//CCW
	float scaleFactor = 0.0;			//<1 : get bigger ; >1 : get smaller
	int trapezoidFactor = 0;
	int camIndex = 0;
	NoteTimeQueue timeQueues[NUMBEROFTIMEQUEUE];
	float fxDecay[NUMBEROFTIMEQUEUE];
	int timeQueueStartIndexArr[NUMBEROFTIMEQUEUE];
};

__device__ NoteTimeQueue timeQueuesOnDevice[NUMBEROFTIMEQUEUE];

Lab1VideoGenerator::Lab1VideoGenerator(): impl(new Impl) {
	//init all members
	for(int i = 0;i<NUMBEROFTIMEQUEUE;i++){
		impl->timeQueues[i].laneIndex = i;
		impl->timeQueues[i].endIndex = 0;
		impl->fxDecay[i] = 0;
		impl->timeQueueStartIndexArr[i] = 0;
	}

	memcpy(impl->timeQueues[0].noteOnTimeQueue, LANE1NOTEONTIMEARR, sizeof(int) * TIMEQUEUEMAXSIZE);
	memcpy(impl->timeQueues[1].noteOnTimeQueue, LANE2NOTEONTIMEARR, sizeof(int) * TIMEQUEUEMAXSIZE);
	memcpy(impl->timeQueues[2].noteOnTimeQueue, LANE3NOTEONTIMEARR, sizeof(int) * TIMEQUEUEMAXSIZE);
	memcpy(impl->timeQueues[3].noteOnTimeQueue, LANE4NOTEONTIMEARR, sizeof(int) * TIMEQUEUEMAXSIZE);
	memcpy(impl->timeQueues[4].noteOnTimeQueue, LANE5NOTEONTIMEARR, sizeof(int) * TIMEQUEUEMAXSIZE);
	memcpy(impl->timeQueues[5].noteOnTimeQueue, LANE6NOTEONTIMEARR, sizeof(int) * TIMEQUEUEMAXSIZE);
	memcpy(impl->timeQueues[0].noteDurationTimeQueue, LANE1NOTEDURATIONTIMEARR, sizeof(int) * TIMEQUEUEMAXSIZE);
	memcpy(impl->timeQueues[1].noteDurationTimeQueue, LANE2NOTEDURATIONTIMEARR, sizeof(int) * TIMEQUEUEMAXSIZE);
	memcpy(impl->timeQueues[2].noteDurationTimeQueue, LANE3NOTEDURATIONTIMEARR, sizeof(int) * TIMEQUEUEMAXSIZE);
	memcpy(impl->timeQueues[3].noteDurationTimeQueue, LANE4NOTEDURATIONTIMEARR, sizeof(int) * TIMEQUEUEMAXSIZE);
	memcpy(impl->timeQueues[4].noteDurationTimeQueue, LANE5NOTEDURATIONTIMEARR, sizeof(int) * TIMEQUEUEMAXSIZE);
	memcpy(impl->timeQueues[5].noteDurationTimeQueue, LANE6NOTEDURATIONTIMEARR, sizeof(int) * TIMEQUEUEMAXSIZE);

	//update end indices
	for(int i = 0;i<6;i++){
		int curEndIndex = 1;
		for(;impl->timeQueues[i].noteOnTimeQueue[curEndIndex] > 0;curEndIndex++);
		impl->timeQueues[i].endIndex = curEndIndex;
	}

	hipMemcpyToSymbol(HIP_SYMBOL(timeQueuesOnDevice), impl->timeQueues, sizeof(NoteTimeQueue) * NUMBEROFTIMEQUEUE);
}

Lab1VideoGenerator::~Lab1VideoGenerator() {}

void Lab1VideoGenerator::get_info(Lab1VideoInfo &info) {
	info.w = W;
	info.h = H;
	info.n_frame = NFRAME;
	// fps = 24/1 = 24
	info.fps_n = 60;
	info.fps_d = 1;
};

__device__ uint3 operator+(const uint3 &a, const uint3 &b){
	return make_uint3(a.x + b.x, a.y + b.y, a.z + b.z);
}

__device__ uint3 operator*(const float &a, const uint3 &b){
	return make_uint3(__float2uint_rd(__uint2float_rn(b.x) * a), __float2uint_rd(__uint2float_rn(b.y) * a), __float2uint_rd(__uint2float_rn(b.z) * a));
}

__device__ uint3 blendRGB(uint3 newRGB, uint3 oldRGB, float alpha){
	return alpha * newRGB + (1.0 - alpha) * oldRGB;
}

__device__ uint3 RGB2YUV(uint3 rgb){
	uint3 tmpYUV;
	tmpYUV.x = __float2uint_rd(0.299*__uint2float_rn(rgb.x) + 0.587*__uint2float_rn(rgb.y) + 0.114*__uint2float_rn(rgb.z));
	tmpYUV.y = __float2uint_rd(-0.169*__uint2float_rn(rgb.x) - 0.331*__uint2float_rn(rgb.y) + 0.5*__uint2float_rn(rgb.z) + 128.0);
	tmpYUV.z = __float2uint_rd(0.5*__uint2float_rn(rgb.x) - 0.419*__uint2float_rn(rgb.y) - 0.081*__uint2float_rn(rgb.z) + 128.0);
	return tmpYUV;
}

__global__ void Draw(uint8_t *yuv) {
	// TODO: draw more complex things here
	// Do not just submit the original file provided by the TA!
	int y = blockIdx.y * blockDim.y + threadIdx.y;
	int x = blockIdx.x * blockDim.x + threadIdx.x;
	if (y < H and x < W) {
		//Set coordinate system to mid-point of screen
		const int realX = x;
		const int realY = y;

		int tmpX = x - W/2;
		int tmpY = y - H/2;

		//Scale
		tmpX = __float2int_rd(__int2float_rd(tmpX) * scaleFactorOnDevice[0]);
		//don't scale Y

		//Rotate
		x = __float2int_rd(__int2float_rd(tmpX) * cos(rotateAngleOnDevice[0]) - __int2float_rd(tmpY) * sin(rotateAngleOnDevice[0]));
		y = __float2int_rd(__int2float_rd(tmpY) * cos(rotateAngleOnDevice[0]) + __int2float_rd(tmpX) * sin(rotateAngleOnDevice[0]));

		//Reset coordinate system
		x += W/2;
		y += H/2;

		uint3 tmpRGB;
		tmpRGB.x = 0;
		tmpRGB.y = 0;
		tmpRGB.z = 0;
		float yProp = __int2float_rd(y + trapezoidFactorOnDevice[0])/__int2float_rd(H + trapezoidFactorOnDevice[0]);
		int3 xRange = make_int3(max(LANE1START, LANEMID - __float2int_rd(yProp * (LANEMID - LANE1START))), min(LANE4END, __float2uint_rd(yProp * (LANE4END - LANEMID) + LANEMID)), 0);
		xRange.z = xRange.y - xRange.x;
		
		if(x > CRITICALLINEXLEFT and x < CRITICALLINEXRIGHT and (H - y) > CRITICALLINEYBOTTOM and (H - y) < CRITICALLINEYTOP){
			tmpRGB.x = 243;
			tmpRGB.y = 253;
			tmpRGB.z = 155;
		}else if(y < H and x < W and y > H / 4 and (H - y) > CRITICALLINEYBOTTOM){
			if(x > xRange.x and x < xRange.y){
				//0: BT; 1: FX
				int2 locatedTimeQueueIndex = make_int2(min(3, __float2int_rd(__int2float_rd(x - xRange.x)/__int2float_rd(xRange.z) * 4.0)), min(5, __float2int_rd(__int2float_rd(x - xRange.x)/__int2float_rd(xRange.z)*2.0) + 4));
				//Fx Notes
				if(timeQueuesOnDevice[locatedTimeQueueIndex.y].isInsideNote(H - y - CRITICALLINEYTOP)){
					tmpRGB.x = 255;
					tmpRGB.y = 137;
					tmpRGB.z = 0;
				}
				//Bt Notes
				if(timeQueuesOnDevice[locatedTimeQueueIndex.x].isInsideNote(H - y - CRITICALLINEYTOP)){
					tmpRGB.x = 255;
					tmpRGB.y = 255;
					tmpRGB.z = 255;
				}
				//Press fx
				tmpRGB = blendRGB(make_uint3(255, 255, 0), tmpRGB, max(fxDecayOnDevice[locatedTimeQueueIndex.x], fxDecayOnDevice[locatedTimeQueueIndex.y]) * max(__int2float_rd(H / 5 - (H - y - CRITICALLINEYTOP)) / __int2float_rd(H/5), 0.0));
				//lane
				if((x - xRange.x) / (xRange.z / 4) < 3 and abs(((x - xRange.x) % (xRange.z / 4)) - (xRange.z / 4)) < 3){
					tmpRGB.x = 225;
					tmpRGB.y = 225;
					tmpRGB.z = 225;
				}
			}else if(x > xRange.x - 6 and x < xRange.x){
				tmpRGB.x = 0;
				tmpRGB.y = 162;
				tmpRGB.z = 255;
			}else if(x > xRange.y and x < xRange.y + 6){
				tmpRGB.x = 255;
				tmpRGB.y = 0;
				tmpRGB.z = 51;
			}
		}


		uint3 returnYUV = RGB2YUV(tmpRGB);
		yuv[realY*W+realX] = returnYUV.x;
		if(realX % 2 == 0 && realY % 2 == 0){
			yuv[W*H + realY*W/4 + realX/2] = returnYUV.y;
			yuv[W*H + W*H/4 + realY*W/4 + realX/2] = returnYUV.z;
		}

	}
}

void Lab1VideoGenerator::Generate(uint8_t *yuv) {
	int curMS = (int)((float)impl->t / 60.0 * 1000.0);
	//Refreshing the camera array index
	if(CAMTIMEARR[impl->camIndex + 1]  <= curMS){
		impl->camIndex++;
	}
	//Refreshing the rotate and scale factor
	impl->rotateAngle = lerp(CAMROTATEARR[impl->camIndex], CAMROTATEARR[impl->camIndex + 1], (float)(curMS - CAMTIMEARR[impl->camIndex]) / (float)(CAMTIMEARR[impl->camIndex + 1] - CAMTIMEARR[impl->camIndex]));
	impl->scaleFactor = lerp(CAMSCALEARR[impl->camIndex], CAMSCALEARR[impl->camIndex + 1], (float)(curMS - CAMTIMEARR[impl->camIndex]) / (float)(CAMTIMEARR[impl->camIndex + 1] - CAMTIMEARR[impl->camIndex]));
	impl->trapezoidFactor = (int)lerp((float)CAMTRAPEZOIDFACTORARR[impl->camIndex], (float)CAMTRAPEZOIDFACTORARR[impl->camIndex + 1], (float)(curMS - CAMTIMEARR[impl->camIndex]) / (float)(CAMTIMEARR[impl->camIndex + 1] - CAMTIMEARR[impl->camIndex]));

	hipMemcpyToSymbol(HIP_SYMBOL(noOfFrameOnDevice), &impl->t, sizeof(int));
	hipMemcpyToSymbol(HIP_SYMBOL(rotateAngleOnDevice), &impl->rotateAngle, sizeof(float));
	hipMemcpyToSymbol(HIP_SYMBOL(scaleFactorOnDevice), &impl->scaleFactor, sizeof(float));
	hipMemcpyToSymbol(HIP_SYMBOL(trapezoidFactorOnDevice), &impl->trapezoidFactor, sizeof(int));
	hipMemcpyToSymbol(HIP_SYMBOL(fxDecayOnDevice), impl->fxDecay, sizeof(float) * NUMBEROFTIMEQUEUE);
	hipMemcpyToSymbol(HIP_SYMBOL(timeQueueStartIndexArrOnDevice), impl->timeQueueStartIndexArr, sizeof(int) * NUMBEROFTIMEQUEUE);
	hipMemset(yuv, 0, W*H);
	hipMemset(yuv+W*H, 128, W*H/2);
	// if(impl->t == 100){
	Draw<<<dim3((W-1)/16+1,(H-1)/12+1), dim3(16,12)>>>(yuv);
	// }
	//Decay Fx
	for(int i = 0;i<NUMBEROFTIMEQUEUE;i++){
		impl->fxDecay[i] = max(impl->fxDecay[i] - 0.08, 0.0);
		if(impl->timeQueues[i].noteOnTimeQueue[impl->timeQueueStartIndexArr[i]] <= curMS){
			impl->fxDecay[i] = 1;
			if(impl->timeQueues[i].noteOnTimeQueue[impl->timeQueueStartIndexArr[i]] + impl->timeQueues[i].noteDurationTimeQueue[impl->timeQueueStartIndexArr[i]] <= curMS){
				impl->timeQueueStartIndexArr[i]++;
			}
		}
	}

	++(impl->t);
}
