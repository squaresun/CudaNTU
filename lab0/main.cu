#include "hip/hip_runtime.h"
#include <cstdio>
#include <cstdlib>
#include "SyncedMemory.h"

#define CHECK {\
	auto e = hipDeviceSynchronize();\
	if (e != hipSuccess) {\
		printf("At " __FILE__ ":%d, %s\n", __LINE__, hipGetErrorString(e));\
		abort();\
	}\
}

const int W = 40;
const int H = 12;

__global__ void Draw(char *frame) {
	// TODO: draw more complex things here
	// Do not just submit the original file provided by the TA!
	const int y = blockIdx.y * blockDim.y + threadIdx.y;
	const int x = blockIdx.x * blockDim.x + threadIdx.x;
	if (y < H && x < W) {
		char c;
		//Drawing boundary
		if (x == W - 1) {
			c = y == H - 1 ? '\0' : '\n';
		}
		else if (y == 0 || y == H - 1 || x == 0 || x == W - 2) {
			c = ':';
		}
		else {
			c = ' ';
		}
		//Drawing pole
		if (x == W - 7) {
			if (y > 4 && y < 10) {
				c = '|';
			}
			else if (y == 10) {
				c = '#';
			}
		}
		//Drawing flag
		if (x == W - 8 && y == 5) {
			c = '<';
		}
		//Drawing trapezoid
		if (y > 4 && y < H - 1 && x < 22) {
			if (x > 17 - 2 * (y - 5)) {
				c = '#';
			}
		}
		frame[y*W + x] = c;
	}
}

int main(int argc, char **argv)
{
	MemoryBuffer<char> frame(W*H);
	auto frame_smem = frame.CreateSync(W*H);
	CHECK;

	Draw << <dim3((W - 1) / 16 + 1, (H - 1) / 12 + 1), dim3(16, 12) >> >(frame_smem.get_gpu_wo());
	CHECK;

	puts(frame_smem.get_cpu_ro());
	CHECK;

	return 0;
}